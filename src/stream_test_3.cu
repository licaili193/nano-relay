
#include <atomic>
#include <chrono>
#include <memory>
#include <thread>

#include <stdio.h>
#include <unistd.h>
#include <sys/ioctl.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <stdlib.h>
#include <signal.h>
#include <poll.h>

#include "NvUtils.h"
#include "nvbuf_utils.h"
#include "NvEglRenderer.h"

#include <glog/logging.h>
#include <gflags/gflags.h>

#include <EGL/egl.h>
#include <EGL/eglext.h>
#include <GLES2/gl2.h>
#include <GLES2/gl2ext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cudaEGL.h"

#include "nvmpi.h"
#include "udp_send_socket.h"
#include "image_processing.h"

#include "camera_grabber.h"

// TODO: clean the include files

DEFINE_string(foreign_addr, "127.0.0.1", "Foreign address");
DEFINE_int32(foreign_port, 6000, "Foreign port");
DEFINE_string(camera, "/dev/video0", "Camera name");

constexpr int width = 640;
constexpr int height = 480;
constexpr size_t bitrate = 800000;
constexpr size_t idr_interval = 10;
constexpr float framerate = 30.f;

std::atomic_bool running;

void myHandler(int s){
  running.store(false);
}

int main(int argc, char** argv) {
  google::InitGoogleLogging(argv[0]);
  google::ParseCommandLineFlags(&argc, &argv, true);
  
  struct sigaction sig_action;
  struct pollfd fds[1];
  NvBufferTransformParams transParams;

  struct sigaction sigIntHandler;
  sigIntHandler.sa_handler = myHandler;
  sigemptyset(&sigIntHandler.sa_mask);
  sigIntHandler.sa_flags = 0;
  sigaction(SIGINT, &sigIntHandler, NULL);

  // Create codec
  nvEncParam param;
  param.width = width;
  param.height = height;
  param.profile = 0; // V4L2_MPEG_VIDEO_H265_PROFILE_MAIN
  param.level = 3; // V4L2_MPEG_VIDEO_H265_LEVEL_2_0_HIGH_TIER
  param.bitrate = bitrate;
  param.peak_bitrate = bitrate;
  param.enableLossless = 0;
  param.mode_vbr = 0; // V4L2_MPEG_VIDEO_BITRATE_MODE_CBR
  param.insert_spspps_idr = 1;
  param.iframe_interval = idr_interval;
  param.idr_interval = idr_interval;
  param.fps_n = static_cast<unsigned int>(framerate);
  param.fps_d = 1;
  param.capture_num = 1;
  param.max_b_frames = 0;
  param.refs = 0;
  param.qmax = -1;
  param.qmin = -1;
  param.hw_preset_type = 1; // V4L2_ENC_HW_PRESET_ULTRAFAST
  param.use_extend_color_format = false;
  param.vbv_size = 0;

  nvmpictx* nvm_ctx = nvmpi_create_encoder(NV_VIDEO_CodingHEVC, &param);
  nvPacket packet;

  // Create socket
  std::unique_ptr<relay::communication::UDPSendSocket> modi_sock =  
      std::unique_ptr<relay::communication::UDPSendSocket>(
          new relay::communication::UDPSendSocket(0, 10, FLAGS_foreign_addr, FLAGS_foreign_port));

  auto d_img_yuyv = image_processing::allocateImageYUYV(width, height);
  CHECK(d_img_yuyv);
  auto d_img_yuv420 = image_processing::allocateImageYUV(width, height);
  CHECK(d_img_yuv420);
  auto d_img_yuv420_s = image_processing::allocateImageYUV(width, height);
  CHECK(d_img_yuv420_s);
  unsigned char* img_yuv420 = new unsigned char[width * height * 3 / 2];
  CHECK(img_yuv420);

  CameraGrabber grabber(d_img_yuyv, FLAGS_camera, width, height);
  
  running.store(true);
  grabber.startStream();

  /* Wait for camera event with timeout = 5000 ms */
  while (running.load()) {
    if (!grabber.newImage()) {
      continue;
    }

    image_processing::yuyv2YUV(width, height, d_img_yuyv, d_img_yuv420);
    image_processing::shuffleYUV(width, height, d_img_yuv420, d_img_yuv420_s);
    image_processing::downloadImageYUV(width, height, d_img_yuv420_s, img_yuv420);

    // Encode frame
    nvFrame frame;
    memset(&frame, 0, sizeof(nvFrame));
    frame.payload[0] = img_yuv420;
    frame.payload_size[0] = width * height;
    frame.payload[1] = img_yuv420 + width * height; 
    frame.payload_size[1] = width * height / 4;
    frame.payload[2] = img_yuv420 + width * height * 5 /4; 
    frame.payload_size[2] = width * height / 4;    
    frame.flags = 0;
    frame.type = NV_PIX_YUV420;
    frame.width = width;
    frame.height = height;

    int ret = nvmpi_encoder_put_frame(nvm_ctx, &frame);

    while (true) {
      memset(&packet, 0, sizeof(nvPacket));
      if (nvmpi_encoder_get_packet(nvm_ctx, &packet) == 0) {
        size_t result_size = packet.payload_size;
        modi_sock->push(result_size, reinterpret_cast<char*>(packet.payload));
      } else {
        break;
      }
    }
  }

  grabber.stopStream();
  hipFree(d_img_yuyv);
  hipFree(d_img_yuv420);
  hipFree(d_img_yuv420_s);
  delete[] img_yuv420;

  LOG(INFO) << "Finished cleanly";

  return 0;
}